#include "hip/hip_runtime.h"
#include <fstream>
#include <algorithm>
#include "ebt/ebt.h"
#include "autodiff/autodiff.h"
#include "la/la.h"
#include "opt/opt.h"
#include "nn/nn.h"

#include "la/la-gpu.h"
#include "autodiff/autodiff-gpu.h"
#include "nn/nn-gpu.h"

#if 0
#include <hip/hip_runtime_api.h>
#endif

struct learning_env {
    std::ifstream input_list;

    std::unordered_map<std::string, int> label_id;
    std::vector<std::string> labels;

    nn::gpu::param_t param;
    nn::gpu::opt_t opt_data;

    double step_size;

    std::string output_param;
    std::string output_opt_data;

    std::unordered_map<std::string, std::string> args;

    learning_env(std::unordered_map<std::string, std::string> args);

    void run();
};

int main(int argc, char *argv[])
{
    ebt::ArgumentSpec spec {
        "learn",
        "Train a feed-forward neural network",
        {
            {"input-list", "", true},
            {"label", "", true},
            {"param", "", true},
            {"opt-data", "", true},
            {"step-size", "", true},
            {"output-param", "", true},
            {"output-opt-data", "", true}
        }
    };

    if (argc == 1) {
        ebt::usage(spec);
        exit(1);
    }

    std::unordered_map<std::string, std::string> args = ebt::parse_args(argc, argv, spec);

    learning_env env { args };
    env.run();

    return 0;
}

learning_env::learning_env(std::unordered_map<std::string, std::string> args)
    : args(args)
{
    input_list.open(args.at("input-list"));

    std::tie(label_id, labels) = nn::load_label_map(args.at("label"));

    param = nn::gpu::param_t(nn::load_param(args.at("param")));
    opt_data = nn::gpu::opt_t(nn::load_opt(args.at("opt-data")));

    step_size = std::stod(args.at("step-size"));

    output_param = args.at("output-param");
    output_opt_data = args.at("output-opt-data");
}

void learning_env::run()
{
    std::string line;

    int sample = 0;
    double loss_mean = 0;
    double loss_var = 0;

    nn::nn_t nn = nn::gpu::make_nn(param);

    while (std::getline(input_list, line)) {
        std::vector<std::string> parts = ebt::split(line);

        std::vector<double> input;
        input.resize(parts.size() - 1);

        std::transform(parts.begin() + 1, parts.end(), input.begin(),
            [](std::string const& s) { return std::stod(s); });

        std::string label = parts.front();

        la::vector<double> gold;
        gold.resize(label_id.size());
        gold(label_id[label]) = 1;

        if (nn.hidden[0]->output == nullptr) {
            nn.hidden[0]->output = std::make_shared<la::gpu::vector<double>>(
                la::gpu::vector<double>(la::vector<double>(input)));
        } else {
            auto& v = autodiff::get_output<la::gpu::vector_like<double>>(nn.hidden[0]);
            la::gpu::to_device(v, la::vector<double>(input));
        }
        autodiff::eval(nn.output, autodiff::gpu::eval_funcs);
        nn::gpu::log_loss loss { autodiff::get_output<la::gpu::vector<double>>(nn.output),
            la::gpu::vector<double>(gold) };

        loss_mean += loss.loss();
        loss_var += std::pow(loss.loss(), 2);

#if DEBUG
        {
            auto& tmp = param.weight[0](0, 0);
            double backup = tmp;
            tmp += 1e-8;

            nn::nn_t nn = nn::make_nn(param);

            nn.hidden[0]->output = std::make_shared<la::gpu::vector<double>>(
                la::gpu::vector<double>(la::vector<double>(input)));
            autodiff::eval(nn.output, autodiff::gpu::eval_funcs);
            nn::gpu::log_loss loss2 { autodiff::get_output<la::gpu::vector<double>>(nn.output), gold };

            std::cout << "numerical grad: " << (loss2.loss() - loss.loss()) / 1e-8 << std::endl;
            tmp = backup;
        }
#endif

        nn.output->grad = std::make_shared<la::gpu::vector<double>>(
            la::gpu::vector<double>(loss.grad()));
        autodiff::grad(nn.output, autodiff::gpu::grad_funcs);

#if DEBUG
        std::cout << "calc grad: "
            << to_host(autodiff::get_grad<la::gpu::matrix<double>>(nn.weight[0]))(0, 0)
            << std::endl;
#endif

        nn::gpu::param_t grad;
        grad.weight.resize(nn.weight.size());
        grad.bias.resize(nn.bias.size());
        nn::gpu::move_grad(grad, nn);
        nn::gpu::move_param(param, nn);

        nn::gpu::adagrad_update(param, grad, opt_data, step_size);

        nn::gpu::move_grad(nn, grad);
        nn::gpu::move_param(nn, param);
        nn::gpu::zero_grad(nn);

        if (sample % 100 == 0) {
            std::cout << "last 100 avg loss: " << loss_mean / 100
                << " var: " << loss_var / 100 - std::pow(loss_mean / 100, 2) << std::endl;
            loss_mean = 0;
            loss_var = 0;
        }

        ++sample;
    }

    nn::gpu::move_param(param, nn);
    save_param(nn::gpu::to_host(param), output_param);
    save_opt(nn::gpu::to_host(opt_data), output_opt_data);
}
