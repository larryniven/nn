#include "nn/nn-gpu.h"
#include "opt/opt-gpu.h"

namespace nn {

    namespace gpu {

        param_t::param_t()
        {}

        param_t::param_t(nn::param_t p)
        {
            for (auto& w: p.weight) {
                weight.push_back(la::gpu::matrix<double>(w));
            }

            for (auto& b: p.bias) {
                bias.push_back(la::gpu::vector<double>(b));
            }

            label_weight = la::gpu::matrix<double>(p.label_weight);
            label_bias = la::gpu::vector<double>(p.label_bias);
        }

        nn::param_t to_host(param_t const& p)
        {
            nn::param_t result;

            for (auto& w: p.weight) {
                result.weight.push_back(to_host(w));
            }

            for (auto& b: p.bias) {
                result.bias.push_back(to_host(b));
            }

            result.label_weight = to_host(p.label_weight);
            result.label_bias = to_host(p.label_bias);

            return result;
        }

        void iadd(param_t& p, param_t const& q)
        {
            for (int i = 0; i < p.weight.size(); ++i) {
                la::gpu::iadd(p.weight[i], q.weight[i]);
                la::gpu::iadd(p.bias[i], q.bias[i]);
            }
            la::gpu::iadd(p.label_weight, q.label_weight);
            la::gpu::iadd(p.label_bias, q.label_bias);
        }

        void resize_as(param_t& p, param_t const& q)
        {
            for (int i = 0; i < q.weight.size(); ++i) {
                la::gpu::matrix<double> m;
                m.resize(q.weight[i].rows(), q.weight[i].cols());
                p.weight.push_back(std::move(m));
            }

            for (int i = 0; i < q.bias.size(); ++i) {
                la::gpu::vector<double> v;
                v.resize(q.bias[i].size());
                p.bias.push_back(std::move(v));
            }

            p.label_weight.resize(q.label_weight.rows(), q.label_weight.cols());

            p.label_bias.resize(q.label_bias.size());
        }

        void zero_param(param_t& p)
        {
            for (int i = 0; i < p.weight.size(); ++i) {
                hipMemset(p.weight[i].data(), 0,
                    p.weight[i].rows() * p.weight[i].cols() * sizeof(double));
                hipMemset(p.bias[i].data(), 0, p.bias[i].size() * sizeof(double));
            }
            hipMemset(p.label_weight.data(), 0,
                p.label_weight.rows() * p.label_weight.cols() * sizeof(double));
            hipMemset(p.label_bias.data(), 0, p.label_bias.size() * sizeof(double));
        }

        opt_t::opt_t()
        {}

        opt_t::opt_t(nn::opt_t o)
            : time(o.time)
            , first_moment(o.first_moment)
            , second_moment(o.second_moment)
        {}

        nn::opt_t to_host(opt_t const& o)
        {
            nn::opt_t result;

            result.time = o.time;
            result.first_moment = to_host(o.first_moment);
            result.second_moment = to_host(o.second_moment);

            return result;
        }

        nn_t make_nn(param_t const& p)
        {
            nn_t nn;

            nn.hidden.push_back(nn.graph.var());

            for (int i = 0; i < p.weight.size(); ++i) {
                auto w_var = nn.graph.var(la::gpu::matrix<double>(p.weight[i]));
                auto b_var = nn.graph.var(la::gpu::vector<double>(p.bias[i]));
                nn.weight.push_back(w_var);
                nn.bias.push_back(b_var);
                nn.hidden.push_back(autodiff::relu(
                    autodiff::add(autodiff::mul(w_var, nn.hidden.back()), b_var)
                ));
            }

            nn.label_weight = nn.graph.var(la::gpu::matrix<double>(p.label_weight));
            nn.label_bias = nn.graph.var(la::gpu::vector<double>(p.label_bias));

            nn.output = autodiff::logsoftmax(autodiff::add(
                autodiff::mul(nn.label_weight, nn.hidden.back()), nn.label_bias));

            return nn;
        }

        nn_t make_nn2(param_t const& p)
        {
            nn_t nn;

            nn.hidden.push_back(nn.graph.var());

            for (int i = 0; i < p.weight.size(); ++i) {
                auto w_var = nn.graph.var(la::gpu::matrix<double>(p.weight[i]));
                auto b_var = nn.graph.var(la::gpu::vector<double>(p.bias[i]));
                nn.weight.push_back(w_var);
                nn.bias.push_back(b_var);
                nn.hidden.push_back(autodiff::relu(
                    autodiff::add(autodiff::mul(w_var, nn.hidden.back()), b_var)
                ));
            }

            nn.label_weight = nn.graph.var(la::gpu::matrix<double>(p.label_weight));
            nn.label_bias = nn.graph.var(la::gpu::vector<double>(p.label_bias));

            std::vector<std::shared_ptr<autodiff::op_t>> hiddens {nn.hidden.begin() + 1, nn.hidden.end()};

            nn.output = autodiff::logsoftmax(autodiff::add(
                autodiff::mul(nn.label_weight, autodiff::add(hiddens)), nn.label_bias));

            return nn;
        }

        void adagrad_update(param_t& p, param_t const& grad,
            opt_t& opt_data, double step_size)
        {
            for (int i = 0; i < p.weight.size(); ++i) {
                opt::gpu::adagrad_update(p.weight[i], grad.weight[i],
                    opt_data.first_moment.weight[i], step_size);
                opt::gpu::adagrad_update(p.bias[i], grad.bias[i],
                    opt_data.first_moment.bias[i], step_size);
            }

            opt::gpu::adagrad_update(p.label_weight, grad.label_weight,
                opt_data.first_moment.label_weight, step_size);
            opt::gpu::adagrad_update(p.label_bias, grad.label_bias,
                opt_data.first_moment.label_bias, step_size);
        }

        void adam_update(param_t& p, param_t const& grad, opt_t& opt_data, double step_size)
        {
            ++opt_data.time;

            for (int i = 0; i < p.weight.size(); ++i) {
                opt::gpu::adam_update(p.weight[i], grad.weight[i],
                    opt_data.first_moment.weight[i], opt_data.second_moment.weight[i],
                    1 + opt_data.time / 1.0e6, step_size, 0.9, 0.999);
                opt::gpu::adam_update(p.bias[i], grad.bias[i],
                    opt_data.first_moment.bias[i], opt_data.second_moment.bias[i],
                    1 + opt_data.time / 1.0e6, step_size, 0.9, 0.999);
            }

            opt::gpu::adam_update(p.label_weight, grad.label_weight,
                opt_data.first_moment.label_weight, opt_data.second_moment.label_weight,
                1 + opt_data.time / 1.0e6, step_size, 0.9, 0.999);
            opt::gpu::adam_update(p.label_bias, grad.label_bias,
                opt_data.first_moment.label_bias, opt_data.second_moment.label_bias,
                1 + opt_data.time / 1.0e6, step_size, 0.9, 0.999);
        }

        void move_param(param_t& result, nn_t& nn)
        {
            for (int i = 0; i < nn.weight.size(); ++i) {
                result.weight[i] = std::move(
                    autodiff::get_output<la::gpu::matrix<double>>(nn.weight[i]));
                result.bias[i] = std::move(
                    autodiff::get_output<la::gpu::vector<double>>(nn.bias[i]));
            }
    
            result.label_weight = std::move(
                autodiff::get_output<la::gpu::matrix<double>>(nn.label_weight));
            result.label_bias = std::move(
                autodiff::get_output<la::gpu::vector<double>>(nn.label_bias));
        }

        void move_param(nn_t& nn, param_t& p)
        {
            for (int i = 0; i < p.weight.size(); ++i) {
                autodiff::get_output<la::gpu::matrix<double>>(nn.weight[i])
                    = std::move(p.weight[i]);
                autodiff::get_output<la::gpu::vector<double>>(nn.bias[i])
                    = std::move(p.bias[i]);
            }
    
            autodiff::get_output<la::gpu::matrix<double>>(nn.label_weight)
                = std::move(p.label_weight);
            autodiff::get_output<la::gpu::vector<double>>(nn.label_bias)
                = std::move(p.label_bias);
        }

        param_t copy_grad(nn_t const& nn)
        {
            param_t result;
    
            for (int i = 0; i < nn.weight.size(); ++i) {
                result.weight.push_back(autodiff::get_grad<la::gpu::matrix<double>>(nn.weight[i]));
                result.bias.push_back(autodiff::get_grad<la::gpu::vector<double>>(nn.bias[i]));
            }
    
            result.label_weight = autodiff::get_grad<la::gpu::matrix<double>>(nn.label_weight);
            result.label_bias = autodiff::get_grad<la::gpu::vector<double>>(nn.label_bias);
    
            return result;
        }

        void move_grad(param_t& result, nn_t const& nn)
        {
            for (int i = 0; i < nn.weight.size(); ++i) {
                result.weight[i] = std::move(
                    autodiff::get_grad<la::gpu::matrix<double>>(nn.weight[i]));
                result.bias[i] = std::move(
                    autodiff::get_grad<la::gpu::vector<double>>(nn.bias[i]));
            }
    
            result.label_weight = std::move(
                autodiff::get_grad<la::gpu::matrix<double>>(nn.label_weight));
            result.label_bias = std::move(
                autodiff::get_grad<la::gpu::vector<double>>(nn.label_bias));
        }

        void move_grad(nn_t& nn, param_t& p)
        {
            for (int i = 0; i < p.weight.size(); ++i) {
                autodiff::get_grad<la::gpu::matrix<double>>(nn.weight[i])
                    = std::move(p.weight[i]);
                autodiff::get_grad<la::gpu::vector<double>>(nn.bias[i])
                    = std::move(p.bias[i]);
            }
    
            autodiff::get_grad<la::gpu::matrix<double>>(nn.label_weight)
                = std::move(p.label_weight);
            autodiff::get_grad<la::gpu::vector<double>>(nn.label_bias)
                = std::move(p.label_bias);
        }

        void zero_grad(nn_t& nn)
        {
            la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(nn.hidden.front()));

            for (int i = 1; i < nn.hidden.size(); ++i) {
                la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(nn.hidden[i]));

                auto add = autodiff::get_child(nn.hidden[i], 0);
                la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(add));

                auto mul = autodiff::get_child(add, 0);
                la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(mul));

                auto bias = autodiff::get_child(add, 1);
                la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(bias));

                auto weight = autodiff::get_child(mul, 0);
                la::gpu::zero(autodiff::get_grad<la::gpu::matrix<double>>(weight));
            }

            la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(nn.output));

            auto add = autodiff::get_child(nn.output, 0);
            la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(add));

            auto mul = autodiff::get_child(add, 0);
            la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(mul));

            auto label_bias = autodiff::get_child(add, 1);
            la::gpu::zero(autodiff::get_grad<la::gpu::vector<double>>(label_bias));

            auto label_weight = autodiff::get_child(mul, 0);
            la::gpu::zero(autodiff::get_grad<la::gpu::matrix<double>>(label_weight));
        }

        log_loss::log_loss(
            la::gpu::vector<double> const& pred, la::gpu::vector<double> const& gold)
            : pred(pred), gold(gold)
        {}
        
        double log_loss::loss()
        {
            return -la::gpu::dot(pred, gold);
        }
        
        la::gpu::vector<double> log_loss::grad()
        {
            return la::gpu::mul(gold, -1);
        }
    }
}
