#include "hip/hip_runtime.h"
#include "la/la.h"
#include "autodiff/autodiff.h"
#include "ebt/ebt.h"
#include "speech/speech.h"
#include <fstream>
#include <vector>
#include "opt/opt.h"
#include "nn/lstm-gpu.h"

struct learning_env {

    std::ifstream frame_batch;
    std::ifstream label_batch;

    lstm::gpu::dblstm_param_t param;
    lstm::gpu::dblstm_param_t opt_data;
    lstm::gpu::dblstm_nn_t nn;

    double step_size;
    double momentum;

    int save_every;

    std::string output_param;
    std::string output_opt_data;

    std::unordered_map<std::string, int> label_id;

    std::unordered_map<std::string, std::string> args;

    learning_env(std::unordered_map<std::string, std::string> args);

    void run();

};

int main(int argc, char *argv[])
{
    ebt::ArgumentSpec spec {
        "learn-lstm-gpu",
        "Train a LSTM frame classifier",
        {
            {"frame-batch", "", true},
            {"label-batch", "", true},
            {"param", "", true},
            {"opt-data", "", true},
            {"step-size", "", true},
            {"momentum", "", false},
            {"save-every", "", false},
            {"output-param", "", false},
            {"output-opt-data", "", false},
            {"label", "", true},
        }
    };

    if (argc == 1) {
        ebt::usage(spec);
        exit(1);
    }

    auto args = ebt::parse_args(argc, argv, spec);

    std::cout << args << std::endl;

    learning_env env { args };

    env.run();

    return 0;
}

learning_env::learning_env(std::unordered_map<std::string, std::string> args)
    : args(args)
{
    frame_batch.open(args.at("frame-batch"));
    label_batch.open(args.at("label-batch"));

    param = lstm::gpu::to_device(
        lstm::load_dblstm_param(args.at("param")));
    opt_data = lstm::gpu::to_device(
        lstm::load_dblstm_param(args.at("opt-data")));

    if (ebt::in(std::string("save-every"), args)) {
        save_every = std::stoi(args.at("save-every"));
    } else {
        save_every = std::numeric_limits<int>::max();
    }

    step_size = std::stod(args.at("step-size"));

    if (ebt::in(std::string("momentum"), args)) {
        momentum = std::stod(args.at("momentum"));
    }

    output_param = "param-last";
    if (ebt::in(std::string("output-param"), args)) {
        output_param = args.at("output-param");
    }

    output_opt_data = "opt-data-last";
    if (ebt::in(std::string("output-opt-data"), args)) {
        output_opt_data = args.at("output-opt-data");
    }

    std::vector<std::string> label_vec = speech::load_label_set(args.at("label"));
    for (int i = 0; i < label_vec.size(); ++i) {
        label_id[label_vec[i]] = i;
    }
}

void learning_env::run()
{
    int i = 1;

    lstm::gpu::dblstm_param_t grad;
    lstm::gpu::resize_as(grad, param);

    autodiff::gpu::memory_pool<double> mem { 100000000 };

    while (1) {
        std::vector<std::vector<double>> frames;

        frames = speech::load_frame_batch(frame_batch);

        std::vector<std::string> labels;

        labels = speech::load_label_batch(label_batch);

        if (!frame_batch || !label_batch) {
            break;
        }

        nn = lstm::gpu::make_dblstm_nn(param, mem, frames);

        lstm::gpu::eval(nn);

        double loss_sum = 0;
        double nframes = 0;

        std::vector<double> gold_block;

        for (int t = 0; t < nn.logprob.size(); ++t) {
            std::vector<double> gold;
            gold.resize(label_id.size());
            gold[label_id.at(labels[t])] = 1;

            gold_block.insert(gold_block.end(), gold.begin(), gold.end());
        }

        double *d = mem.alloc(gold_block.size());
        la::gpu::weak_vector<double> gold_device_block { d, gold_block.size() };
        la::gpu::to_device(gold_device_block, la::vector<double>(gold_block));
        unsigned int gold_dim = label_id.size();

        for (int t = 0; t < nn.logprob.size(); ++t) {
            auto& pred = autodiff::get_output<la::gpu::vector_like<double>>(nn.logprob.at(t));

            lstm::gpu::log_loss loss {
                la::gpu::weak_vector<double>(gold_device_block.data() + t * gold_dim, gold_dim),
                pred };

            if (std::isnan(loss.loss())) {
                std::cerr << "loss is nan" << std::endl;
                exit(1);
            } else {
                loss_sum += loss.loss();
                nframes += 1;
            }

            la::gpu::weak_vector<double> g(gold_device_block.data() + t * gold_dim, gold_dim);

            la::gpu::imul(g, -1);

            nn.logprob[t]->grad = std::make_shared<la::gpu::weak_vector<double>>(g);
        }


        lstm::gpu::attach_grad(grad, nn);
        lstm::gpu::grad(nn);

        if (ebt::in(std::string("momentum"), args)) {
            lstm::gpu::const_step_update_momentum(param, grad, opt_data, momentum, step_size);
        } else {
            lstm::gpu::adagrad_update(param, grad, opt_data, step_size);
        }

        lstm::gpu::zero(grad);

        mem.reset();

        std::cout << "loss: " << loss_sum / nframes << std::endl;

#if 0
        {
            lstm::param_t p = param;
            p.hidden_input(0, 0) += 1e-8;
            lstm::nn_t nn2 = lstm::make_nn(p, frames);
            lstm::eval(nn2);
            auto& pred = autodiff::get_output<la::vector_like<double>>(nn2.logprob.at(1));
            la::vector<double> gold;
            gold.resize(label_id.size());
            gold(label_id.at(labels[1])) = 1;
            lstm::log_loss loss2 { gold, pred };

            auto& grad = autodiff::get_grad<la::matrix_like<double>>(nn.hidden_input);
            std::cout << (loss2.loss() - loss_1) / 1e-8 << " " << grad(0, 0) << std::endl;
        }
#endif

#ifdef DEBUG_TOP
        if (i == DEBUG_TOP) {
            break;
        }
#endif

        ++i;
    }

    lstm::save_dblstm_param(lstm::gpu::to_host(param), output_param);
    lstm::save_dblstm_param(lstm::gpu::to_host(opt_data), output_opt_data);
}

