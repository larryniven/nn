#include "hip/hip_runtime.h"
#include "nn/lstm-gpu.h"
#include "opt/opt-gpu.h"
#include "autodiff/autodiff-gpu.h"
#include <algorithm>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>

namespace lstm {

    namespace gpu {

        lstm::lstm_feat_param_t to_host(lstm_feat_param_t const& param)
        {
            lstm::lstm_feat_param_t result;

            result.hidden_input = la::gpu::to_host(param.hidden_input);
            result.hidden_output = la::gpu::to_host(param.hidden_output);
            result.hidden_bias = la::gpu::to_host(param.hidden_bias);

            result.input_input = la::gpu::to_host(param.input_input);
            result.input_output = la::gpu::to_host(param.input_output);
            result.input_peep = la::gpu::to_host(param.input_peep);
            result.input_bias = la::gpu::to_host(param.input_bias);

            result.output_input = la::gpu::to_host(param.output_input);
            result.output_output = la::gpu::to_host(param.output_output);
            result.output_peep = la::gpu::to_host(param.output_peep);
            result.output_bias = la::gpu::to_host(param.output_bias);

            result.forget_input = la::gpu::to_host(param.forget_input);
            result.forget_output = la::gpu::to_host(param.forget_output);
            result.forget_peep = la::gpu::to_host(param.forget_peep);
            result.forget_bias = la::gpu::to_host(param.forget_bias);

            return result;
        }

        lstm_feat_param_t to_device(lstm::lstm_feat_param_t const& param)
        {
            lstm_feat_param_t result;

            result.hidden_input = la::gpu::matrix<double>(param.hidden_input);
            result.hidden_output = la::gpu::matrix<double>(param.hidden_output);
            result.hidden_bias = la::gpu::vector<double>(param.hidden_bias);

            result.input_input = la::gpu::matrix<double>(param.input_input);
            result.input_output = la::gpu::matrix<double>(param.input_output);
            result.input_peep = la::gpu::vector<double>(param.input_peep);
            result.input_bias = la::gpu::vector<double>(param.input_bias);

            result.output_input = la::gpu::matrix<double>(param.output_input);
            result.output_output = la::gpu::matrix<double>(param.output_output);
            result.output_peep = la::gpu::vector<double>(param.output_peep);
            result.output_bias = la::gpu::vector<double>(param.output_bias);

            result.forget_input = la::gpu::matrix<double>(param.forget_input);
            result.forget_output = la::gpu::matrix<double>(param.forget_output);
            result.forget_peep = la::gpu::vector<double>(param.forget_peep);
            result.forget_bias = la::gpu::vector<double>(param.forget_bias);

            return result;
        }

        void resize_as(lstm_feat_param_t& a, lstm_feat_param_t const& b)
        {
            a.hidden_input.resize(b.hidden_input.rows(), b.hidden_input.cols());
            a.hidden_output.resize(b.hidden_output.rows(), b.hidden_output.cols());
            a.hidden_bias.resize(b.hidden_bias.size());

            a.input_input.resize(b.input_input.rows(), b.input_input.cols());
            a.input_output.resize(b.input_output.rows(), b.input_output.cols());
            a.input_peep.resize(b.input_peep.size());
            a.input_bias.resize(b.input_bias.size());

            a.output_input.resize(b.output_input.rows(), b.output_input.cols());
            a.output_output.resize(b.output_output.rows(), b.output_output.cols());
            a.output_peep.resize(b.output_peep.size());
            a.output_bias.resize(b.output_bias.size());

            a.forget_input.resize(b.forget_input.rows(), b.forget_input.cols());
            a.forget_output.resize(b.forget_output.rows(), b.forget_output.cols());
            a.forget_peep.resize(b.forget_peep.size());
            a.forget_bias.resize(b.forget_bias.size());
        }

        void zero(lstm_feat_param_t& p)
        {
            la::gpu::zero(p.hidden_input);
            la::gpu::zero(p.hidden_output);
            la::gpu::zero(p.hidden_bias);

            la::gpu::zero(p.input_input);
            la::gpu::zero(p.input_output);
            la::gpu::zero(p.input_peep);
            la::gpu::zero(p.input_bias);

            la::gpu::zero(p.output_input);
            la::gpu::zero(p.output_output);
            la::gpu::zero(p.output_peep);
            la::gpu::zero(p.output_bias);

            la::gpu::zero(p.forget_input);
            la::gpu::zero(p.forget_output);
            la::gpu::zero(p.forget_peep);
            la::gpu::zero(p.forget_bias);
        }

        void adagrad_update(lstm_feat_param_t& p, lstm_feat_param_t const& grad,
            lstm_feat_param_t& opt_data, double step_size)
        {
            opt::gpu::adagrad_update(p.hidden_input, grad.hidden_input,
                opt_data.hidden_input, step_size);
            opt::gpu::adagrad_update(p.hidden_output, grad.hidden_output,
                opt_data.hidden_output, step_size);
            opt::gpu::adagrad_update(p.hidden_bias, grad.hidden_bias,
                opt_data.hidden_bias, step_size);

            opt::gpu::adagrad_update(p.input_input, grad.input_input,
                opt_data.input_input, step_size);
            opt::gpu::adagrad_update(p.input_output, grad.input_output,
                opt_data.input_output, step_size);
            opt::gpu::adagrad_update(p.input_peep, grad.input_peep,
                opt_data.input_peep, step_size);
            opt::gpu::adagrad_update(p.input_bias, grad.input_bias,
                opt_data.input_bias, step_size);

            opt::gpu::adagrad_update(p.output_input, grad.output_input,
                opt_data.output_input, step_size);
            opt::gpu::adagrad_update(p.output_output, grad.output_output,
                opt_data.output_output, step_size);
            opt::gpu::adagrad_update(p.output_peep, grad.output_peep,
                opt_data.output_peep, step_size);
            opt::gpu::adagrad_update(p.output_bias, grad.output_bias,
                opt_data.output_bias, step_size);

            opt::gpu::adagrad_update(p.forget_input, grad.forget_input,
                opt_data.forget_input, step_size);
            opt::gpu::adagrad_update(p.forget_output, grad.forget_output,
                opt_data.forget_output, step_size);
            opt::gpu::adagrad_update(p.forget_peep, grad.forget_peep,
                opt_data.forget_peep, step_size);
            opt::gpu::adagrad_update(p.forget_bias, grad.forget_bias,
                opt_data.forget_bias, step_size);
        }

        void const_step_update_momentum(lstm_feat_param_t& p, lstm_feat_param_t const& grad,
            lstm_feat_param_t& opt_data, double momentum, double step_size)
        {
            opt::gpu::const_step_update_momentum(p.hidden_input, grad.hidden_input,
                opt_data.hidden_input, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.hidden_output, grad.hidden_output,
                opt_data.hidden_output, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.hidden_bias, grad.hidden_bias,
                opt_data.hidden_bias, momentum, step_size);

            opt::gpu::const_step_update_momentum(p.input_input, grad.input_input,
                opt_data.input_input, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.input_output, grad.input_output,
                opt_data.input_output, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.input_peep, grad.input_peep,
                opt_data.input_peep, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.input_bias, grad.input_bias,
                opt_data.input_bias, momentum, step_size);

            opt::gpu::const_step_update_momentum(p.output_input, grad.output_input,
                opt_data.output_input, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.output_output, grad.output_output,
                opt_data.output_output, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.output_peep, grad.output_peep,
                opt_data.output_peep, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.output_bias, grad.output_bias,
                opt_data.output_bias, momentum, step_size);

            opt::gpu::const_step_update_momentum(p.forget_input, grad.forget_input,
                opt_data.forget_input, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.forget_output, grad.forget_output,
                opt_data.forget_output, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.forget_peep, grad.forget_peep,
                opt_data.forget_peep, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.forget_bias, grad.forget_bias,
                opt_data.forget_bias, momentum, step_size);
        }

        lstm::lstm_feat_nn_t make_forward_lstm_feat_nn(autodiff::computation_graph& g,
            autodiff::gpu::memory_pool<double>& mem,
            lstm_feat_param_t& p,
            std::vector<std::shared_ptr<autodiff::op_t>> const& inputs)
        {
            lstm_feat_nn_t result;

            result.hidden_input = g.var(la::gpu::weak_matrix<double>(p.hidden_input));
            result.hidden_output = g.var(la::gpu::weak_matrix<double>(p.hidden_output));
            result.hidden_bias = g.var(la::gpu::weak_vector<double>(p.hidden_bias));

            result.input_input = g.var(la::gpu::weak_matrix<double>(p.input_input));
            result.input_output = g.var(la::gpu::weak_matrix<double>(p.input_output));
            result.input_peep = g.var(la::gpu::weak_vector<double>(p.input_peep));
            result.input_bias = g.var(la::gpu::weak_vector<double>(p.input_bias));

            result.output_input = g.var(la::gpu::weak_matrix<double>(p.output_input));
            result.output_output = g.var(la::gpu::weak_matrix<double>(p.output_output));
            result.output_peep = g.var(la::gpu::weak_vector<double>(p.output_peep));
            result.output_bias = g.var(la::gpu::weak_vector<double>(p.output_bias));

            result.forget_input = g.var(la::gpu::weak_matrix<double>(p.forget_input));
            result.forget_output = g.var(la::gpu::weak_matrix<double>(p.forget_output));
            result.forget_peep = g.var(la::gpu::weak_vector<double>(p.forget_peep));
            result.forget_bias = g.var(la::gpu::weak_vector<double>(p.forget_bias));

            result.hidden.push_back(autodiff::tanh(
                autodiff::add(autodiff::mul(result.hidden_input, inputs.front()),
                result.hidden_bias)));

            result.input_gate.push_back(autodiff::logistic(
                autodiff::add(autodiff::mul(result.input_input, inputs.front()),
                result.input_bias)));

            result.cell.push_back(autodiff::emul(result.input_gate.back(),
                result.hidden.back()));

            result.output_gate.push_back(autodiff::logistic(autodiff::add(
                std::vector<std::shared_ptr<autodiff::op_t>> {
                    autodiff::mul(result.output_input, inputs.front()),
                    autodiff::emul(result.output_peep, result.cell.back()),
                    result.output_bias
                })));

            result.output.push_back(autodiff::emul(result.output_gate.back(),
                autodiff::tanh(result.cell.back())));

            for (int i = 1; i < inputs.size(); ++i) {
                result.hidden.push_back(autodiff::tanh(autodiff::add(
                    std::vector<std::shared_ptr<autodiff::op_t>> {
                        autodiff::mul(result.hidden_input, inputs[i]),
                        autodiff::mul(result.hidden_output, result.output.back()),
                        result.hidden_bias
                    })));

                result.input_gate.push_back(autodiff::logistic(autodiff::add(
                    std::vector<std::shared_ptr<autodiff::op_t>> {
                        autodiff::mul(result.input_input, inputs[i]),
                        autodiff::mul(result.input_output, result.output.back()),
                        autodiff::emul(result.input_peep, result.cell.back()),
                        result.input_bias
                    })));

                result.forget_gate.push_back(autodiff::logistic(autodiff::add(
                    std::vector<std::shared_ptr<autodiff::op_t>> {
                        autodiff::mul(result.forget_input, inputs[i]),
                        autodiff::mul(result.forget_output, result.output.back()),
                        autodiff::emul(result.forget_peep, result.cell.back()),
                        result.forget_bias
                    })));

                result.cell.push_back(autodiff::add(
                    autodiff::emul(result.forget_gate.back(), result.cell.back()),
                    autodiff::emul(result.input_gate.back(), result.hidden.back())));

                result.output_gate.push_back(autodiff::logistic(autodiff::add(
                    std::vector<std::shared_ptr<autodiff::op_t>> {
                        autodiff::mul(result.output_input, inputs[i]),
                        autodiff::mul(result.output_output, result.output.back()),
                        autodiff::emul(result.output_peep, result.cell.back()),
                        result.output_bias
                    })));

                result.output.push_back(autodiff::emul(result.output_gate.back(),
                    autodiff::tanh(result.cell.back())));
            }

            return result;
        }

        lstm::lstm_feat_nn_t make_backward_lstm_feat_nn(autodiff::computation_graph& g,
            autodiff::gpu::memory_pool<double>& mem,
            lstm_feat_param_t& p,
            std::vector<std::shared_ptr<autodiff::op_t>> const& inputs)
        {
            std::vector<std::shared_ptr<autodiff::op_t>> rev_inputs = inputs;
            std::reverse(rev_inputs.begin(), rev_inputs.end());

            lstm_feat_nn_t result = make_forward_lstm_feat_nn(g, mem, p, rev_inputs);

            std::reverse(result.cell.begin(), result.cell.end());
            std::reverse(result.hidden.begin(), result.hidden.end());
            std::reverse(result.input_gate.begin(), result.input_gate.end());
            std::reverse(result.output_gate.begin(), result.output_gate.end());
            std::reverse(result.forget_gate.begin(), result.forget_gate.end());
            std::reverse(result.output.begin(), result.output.end());

            return result;
        }

        void attach_grad(lstm_feat_param_t& grad, lstm::lstm_feat_nn_t const& nn)
        {
            nn.hidden_input->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.hidden_input));
            nn.hidden_output->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.hidden_output));
            nn.hidden_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.hidden_bias));

            nn.input_input->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.input_input));
            nn.input_output->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.input_output));
            nn.input_peep->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.input_peep));
            nn.input_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.input_bias));

            nn.output_input->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.output_input));
            nn.output_output->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.output_output));
            nn.output_peep->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.output_peep));
            nn.output_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.output_bias));

            nn.forget_input->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.forget_input));
            nn.forget_output->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.forget_output));
            nn.forget_peep->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.forget_peep));
            nn.forget_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.forget_bias));
        }

        lstm::blstm_feat_param_t to_host(blstm_feat_param_t const& param)
        {
            lstm::blstm_feat_param_t result;

            result.forward_param = to_host(param.forward_param);
            result.backward_param = to_host(param.backward_param);

            result.forward_output_weight = la::gpu::to_host(param.forward_output_weight);
            result.backward_output_weight = la::gpu::to_host(param.backward_output_weight);
            result.output_bias = la::gpu::to_host(param.output_bias);

            return result;
        }

        blstm_feat_param_t to_device(lstm::blstm_feat_param_t const& param)
        {
            blstm_feat_param_t result;

            result.forward_param = to_device(param.forward_param);
            result.backward_param = to_device(param.backward_param);

            result.forward_output_weight = la::gpu::matrix<double>(
                param.forward_output_weight);
            result.backward_output_weight = la::gpu::matrix<double>(
                param.backward_output_weight);
            result.output_bias = la::gpu::vector<double>(param.output_bias);

            return result;
        }

        void resize_as(blstm_feat_param_t& a, blstm_feat_param_t const& b)
        {
            resize_as(a.forward_param, b.forward_param);
            resize_as(a.backward_param, b.backward_param);

            a.forward_output_weight.resize(
                b.forward_output_weight.rows(), b.forward_output_weight.cols());
            a.backward_output_weight.resize(
                b.backward_output_weight.rows(), b.backward_output_weight.cols());
            a.output_bias.resize(b.output_bias.size());
        }

        void zero(blstm_feat_param_t& p)
        {
            zero(p.forward_param);
            zero(p.backward_param);

            la::gpu::zero(p.forward_output_weight);
            la::gpu::zero(p.backward_output_weight);
            la::gpu::zero(p.output_bias);
        }

        void adagrad_update(blstm_feat_param_t& p, blstm_feat_param_t const& grad,
            blstm_feat_param_t& opt_data, double step_size)
        {
            adagrad_update(p.forward_param, grad.forward_param,
                opt_data.forward_param, step_size);
            adagrad_update(p.backward_param, grad.backward_param,
                opt_data.backward_param, step_size);

            opt::gpu::adagrad_update(p.forward_output_weight, grad.forward_output_weight,
                opt_data.forward_output_weight, step_size);
            opt::gpu::adagrad_update(p.backward_output_weight, grad.backward_output_weight,
                opt_data.backward_output_weight, step_size);
            opt::gpu::adagrad_update(p.output_bias, grad.output_bias,
                opt_data.output_bias, step_size);
        }

        void const_step_update_momentum(blstm_feat_param_t& p, blstm_feat_param_t const& grad,
            blstm_feat_param_t& opt_data, double momentum, double step_size)
        {
            const_step_update_momentum(p.forward_param, grad.forward_param,
                opt_data.forward_param, momentum, step_size);
            const_step_update_momentum(p.backward_param, grad.backward_param,
                opt_data.backward_param, momentum, step_size);

            opt::gpu::const_step_update_momentum(p.forward_output_weight, grad.forward_output_weight,
                opt_data.forward_output_weight, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.backward_output_weight, grad.backward_output_weight,
                opt_data.backward_output_weight, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.output_bias, grad.output_bias,
                opt_data.output_bias, momentum, step_size);
        }

        lstm::blstm_feat_nn_t make_blstm_feat_nn(autodiff::computation_graph& g,
            autodiff::gpu::memory_pool<double>& mem,
            blstm_feat_param_t& p,
            std::vector<std::shared_ptr<autodiff::op_t>> const& inputs)
        {
            blstm_feat_nn_t result;

            result.forward_feat_nn = make_forward_lstm_feat_nn(g, mem, p.forward_param, inputs);
            result.backward_feat_nn = make_backward_lstm_feat_nn(g, mem, p.backward_param, inputs);

            result.forward_output_weight = g.var(
                la::gpu::weak_matrix<double>(p.forward_output_weight));
            result.backward_output_weight = g.var(
                la::gpu::weak_matrix<double>(p.backward_output_weight));
            result.output_bias = g.var(
                la::gpu::weak_vector<double>(p.output_bias));

            for (int i = 0; i < result.forward_feat_nn.output.size(); ++i) {
                result.output.push_back(autodiff::add(
                    std::vector<std::shared_ptr<autodiff::op_t>> {
                        autodiff::mul(result.forward_output_weight,
                            result.forward_feat_nn.output[i]),
                        autodiff::mul(result.backward_output_weight,
                            result.backward_feat_nn.output[i]),
                        result.output_bias
                    }));
            }

            return result;
        }

        void attach_grad(blstm_feat_param_t& grad, lstm::blstm_feat_nn_t const& nn)
        {
            attach_grad(grad.forward_param, nn.forward_feat_nn);
            attach_grad(grad.backward_param, nn.backward_feat_nn);

            nn.forward_output_weight->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.forward_output_weight));
            nn.backward_output_weight->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.backward_output_weight));
            nn.output_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.output_bias));
        }

        lstm::dblstm_param_t to_host(dblstm_param_t const& param)
        {
            lstm::dblstm_param_t result;

            for (int i = 0; i < param.layer.size(); ++i) {
                result.layer.push_back(to_host(param.layer[i]));
            }

            result.softmax_weight = la::gpu::to_host(param.softmax_weight);
            result.softmax_bias = la::gpu::to_host(param.softmax_bias);

            return result;
        }

        dblstm_param_t to_device(lstm::dblstm_param_t const& param)
        {
            dblstm_param_t result;

            for (int i = 0; i < param.layer.size(); ++i) {
                result.layer.push_back(to_device(param.layer[i]));
            }

            result.softmax_weight = la::gpu::matrix<double>(param.softmax_weight);
            result.softmax_bias = la::gpu::vector<double>(param.softmax_bias);

            return result;
        }

        void resize_as(dblstm_param_t& a, dblstm_param_t const& b)
        {
            a.layer.resize(b.layer.size());

            for (int i = 0; i < b.layer.size(); ++i) {
                resize_as(a.layer[i], b.layer[i]);
            }

            a.softmax_weight.resize(b.softmax_weight.rows(), b.softmax_weight.cols());
            a.softmax_bias.resize(b.softmax_bias.size());
        }

        void zero(dblstm_param_t& p)
        {
            for (auto& ell: p.layer) {
                zero(ell);
            }

            la::gpu::zero(p.softmax_weight);
            la::gpu::zero(p.softmax_bias);
        }

        void adagrad_update(dblstm_param_t& p, dblstm_param_t const& grad,
            dblstm_param_t& opt_data, double step_size)
        {
            for (int i = 0; i < p.layer.size(); ++i) {
                adagrad_update(p.layer[i], grad.layer[i],
                    opt_data.layer[i], step_size);
            }

            opt::gpu::adagrad_update(p.softmax_weight, grad.softmax_weight,
                opt_data.softmax_weight, step_size);
            opt::gpu::adagrad_update(p.softmax_bias, grad.softmax_bias,
                opt_data.softmax_bias, step_size);
        }

        void const_step_update_momentum(dblstm_param_t& p, dblstm_param_t const& grad,
            dblstm_param_t& opt_data, double momentum, double step_size)
        {
            for (int i = 0; i < p.layer.size(); ++i) {
                const_step_update_momentum(p.layer[i], grad.layer[i],
                    opt_data.layer[i], momentum, step_size);
            }

            opt::gpu::const_step_update_momentum(p.softmax_weight, grad.softmax_weight,
                opt_data.softmax_weight, momentum, step_size);
            opt::gpu::const_step_update_momentum(p.softmax_bias, grad.softmax_bias,
                opt_data.softmax_bias, momentum, step_size);
        }

        dblstm_nn_t make_dblstm_nn(dblstm_param_t& p,
            autodiff::gpu::memory_pool<double>& mem,
            std::vector<std::vector<double>> const& frames)
        {
            dblstm_nn_t result;
            result.mem = &mem;

            unsigned int dim = frames.front().size();

            std::vector<double> frame_block;
            for (auto& f: frames) {
                frame_block.insert(frame_block.end(), f.begin(), f.end());
            }

            double *d = result.mem->alloc(frame_block.size());

            la::gpu::weak_vector<double> frame_device_block(d, frame_block.size());
            la::gpu::to_device(frame_device_block, la::vector<double>(frame_block));

            std::vector<std::shared_ptr<autodiff::op_t>> inputs;

            for (int i = 0; i < frames.size(); ++i) {
                auto v = result.graph.var(la::gpu::weak_vector<double>(
                    frame_device_block.data() + i * dim, dim));
                double *d = result.mem->alloc(dim);
                la::gpu::weak_vector<double> g(d, dim);
                la::gpu::zero(g);
                v->grad = std::make_shared<la::gpu::weak_vector<double>>(g);
                
                inputs.push_back(v);
            }

            for (int i = 0; i < p.layer.size(); ++i) {
                if (i == 0) {
                    result.layer.push_back(make_blstm_feat_nn(
                        result.graph, mem, p.layer[0], inputs));
                } else {
                    result.layer.push_back(make_blstm_feat_nn(
                        result.graph, mem, p.layer[i], result.layer[i-1].output));
                }
            }

            result.softmax_weight = result.graph.var(la::gpu::weak_matrix<double>(p.softmax_weight));
            result.softmax_bias = result.graph.var(la::gpu::weak_vector<double>(p.softmax_bias));

            for (int i = 0; i < result.layer.back().output.size(); ++i) {
                result.logprob.push_back(autodiff::logsoftmax(autodiff::add(
                    autodiff::mul(result.softmax_weight, result.layer.back().output[i]),
                    result.softmax_bias)));
            }

            std::vector<std::shared_ptr<autodiff::op_t>> order = autodiff::topo_order(result.logprob);
            autodiff::gpu::alloc(order, mem, autodiff::gpu::alloc_funcs);

            return result;
        }

        void attach_grad(dblstm_param_t& grad, dblstm_nn_t const& nn)
        {
            for (int i = 0; i < nn.layer.size(); ++i) {
                attach_grad(grad.layer[i], nn.layer[i]);
            }

            nn.softmax_weight->grad = std::make_shared<la::gpu::weak_matrix<double>>(
                la::gpu::weak_matrix<double>(grad.softmax_weight));
            nn.softmax_bias->grad = std::make_shared<la::gpu::weak_vector<double>>(
                la::gpu::weak_vector<double>(grad.softmax_bias));
        }

        void eval(dblstm_nn_t const& nn)
        {
            std::vector<std::shared_ptr<autodiff::op_t>> order
                = autodiff::topo_order(nn.logprob);

            autodiff::eval(order, autodiff::gpu::eval_funcs);
        }

        void grad(dblstm_nn_t const& nn)
        {
            std::vector<std::shared_ptr<autodiff::op_t>> order
                = autodiff::topo_order(nn.logprob);

            autodiff::grad(order, autodiff::gpu::grad_funcs);
        }

        double log_loss::loss()
        {
            return -la::gpu::dot(gold, pred);
        }

    }

}
