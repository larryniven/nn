#include "hip/hip_runtime.h"
#include "nn/tensor-tree-gpu.h"
#include "opt/opt.h"
#include "opt/opt-gpu.h"

namespace tensor_tree {

    namespace gpu {

        void to_device(std::shared_ptr<vertex> t)
        {
            auto order = leaves_pre_order(t);

            for (auto& t: order) {
                if (t->data == nullptr) {
                    continue;
                }

                if (t->type == "tensor") {
                    la::gpu::tensor<double> dt { get_tensor(t) };
                    t->type = "gpu-tensor";
                    t->data = std::make_shared<la::gpu::tensor<double>>(std::move(dt));
                }
            }
        }

        void to_host(std::shared_ptr<vertex> t)
        {
            auto order = leaves_pre_order(t);

            for (auto& t: order) {
                if (t->data == nullptr) {
                    continue;
                }

                if (t->type == "gpu-tensor") {
                    t->data = std::make_shared<la::tensor<double>>(la::gpu::to_host(get_gpu_tensor(t)));
                    t->type = "tensor";
                }
            }
        }

        la::gpu::tensor<double>& get_gpu_tensor(std::shared_ptr<vertex> t)
        {
            if (t->type == "gpu-tensor") {
                return get_data<la::gpu::tensor<double>>(t);
            } else {
                throw std::logic_error("expecting gpu-tensor");
            }
        }

        void resize_as(std::shared_ptr<vertex> p1, std::shared_ptr<vertex> p2)
        {
            auto p1_order = leaves_pre_order(p1);
            auto p2_order = leaves_pre_order(p2);

            for (int i = 0; i < p1_order.size(); ++i) {
                if (p1_order[i]->type == "tensor") {
                    la::tensor<double> m;
                    auto& m2 = get_tensor(p2_order[i]);
                    la::resize_as(m, m2);
                    p1_order[i]->data = std::make_shared<la::tensor<double>>(std::move(m));
                } else if (p1_order[i]->type == "gpu-tensor") {
                    la::gpu::tensor<double> m;
                    auto& m2 = get_gpu_tensor(p2_order[i]);
                    la::gpu::resize_as(m, m2);
                    p1_order[i]->data = std::make_shared<la::gpu::tensor<double>>(std::move(m));
                }
            }
        }

        void imul(std::shared_ptr<vertex> root, double a)
        {
            auto order = leaves_pre_order(root);

            for (auto& t: order) {
                if (t->data == nullptr) {
                    continue;
                }

                if (t->type == "tensor") {
                    la::imul(get_tensor(t), a);
                } else if (t->type == "gpu-tensor") {
                    la::gpu::imul(get_gpu_tensor(t), a);
                }
            }
        }

        void iadd(std::shared_ptr<vertex> p1, std::shared_ptr<vertex> p2)
        {
            auto p1_order = leaves_pre_order(p1);
            auto p2_order = leaves_pre_order(p2);

            for (int i = 0; i < p1_order.size(); ++i) {
                if (p2_order[i]->data == nullptr) {
                    continue;
                }

                if (p1_order[i]->type == "tensor") {
                    la::iadd(get_tensor(p1_order[1]), get_tensor(p2_order[i]));
                } else if (p1_order[i]->type == "gpu-tensor") {
                    la::gpu::iadd(get_gpu_tensor(p1_order[1]), get_gpu_tensor(p2_order[i]));
                }
            }
        }

        void isub(std::shared_ptr<vertex> p1, std::shared_ptr<vertex> p2)
        {
            auto p1_order = leaves_pre_order(p1);
            auto p2_order = leaves_pre_order(p2);

            for (int i = 0; i < p1_order.size(); ++i) {
                if (p2_order[i]->data == nullptr) {
                    continue;
                }

                if (p1_order[i]->type == "tensor") {
                    la::isub(get_tensor(p1_order[i]), get_tensor(p2_order[i]));
                } else if (p1_order[i]->type == "gpu-tensor") {
                    la::gpu::isub(get_gpu_tensor(p1_order[i]), get_gpu_tensor(p2_order[i]));
                }
            }
        }

        void zero(std::shared_ptr<vertex> root)
        {
            auto order = leaves_pre_order(root);

            for (auto& t: order) {
                if (t->data == nullptr) {
                    continue;
                }

                if (t->type == "tensor") {
                    la::zero(get_tensor(t));
                } else if (t->type == "gpu-tensor") {
                    la::gpu::zero(get_gpu_tensor(t));
                }
            }
        }

        double norm(std::shared_ptr<vertex> root)
        {
            auto order = leaves_pre_order(root);

            double result = 0;

            for (int i = 0; i < order.size(); ++i) {
                if (order[i]->data == nullptr) {
                    continue;
                }

                if (order[i]->type == "tensor") {
                    result += std::pow(la::norm(get_tensor(order[i])), 2);
                } else if (order[i]->type == "gpu-tensor") {
                    result += std::pow(la::gpu::norm(get_gpu_tensor(order[i])), 2);
                }
            }

            return std::sqrt(result);
        }

        bool has_nan(std::shared_ptr<vertex> root)
        {
            auto order = leaves_pre_order(root);

            for (int i = 0; i < order.size(); ++i) {
                if (order[i]->data == nullptr) {
                    continue;
                }

                if (order[i]->type == "tensor") {
                    if (la::has_nan(get_tensor(order[i]))) {
                        return true;
                    }
                } else if (order[i]->type == "gpu-tensor") {
                    if (la::gpu::has_nan(get_gpu_tensor(order[i]))) {
                        return true;
                    }
                }
            }

            return false;
        }

        void const_step_update(std::shared_ptr<vertex> param, std::shared_ptr<vertex> grad,
            double step_size)
        {
            auto param_order = leaves_pre_order(param);
            auto grad_order = leaves_pre_order(grad);

            assert(param_order.size() == grad_order.size());

            for (int i = 0; i < param_order.size(); ++i) {
                if (grad_order[i]->data == nullptr) {
                    continue;
                }

                if (param_order[i]->type == "tensor") {
                    opt::const_step_update(
                        get_tensor(param_order[i]),
                        get_tensor(grad_order[i]),
                        step_size);
                } else if (param_order[i]->type == "gpu-tensor") {
                    opt::gpu::const_step_update(
                        get_gpu_tensor(param_order[i]),
                        get_gpu_tensor(grad_order[i]),
                        step_size);
                }
            }
        }

        void const_step_update_momentum(std::shared_ptr<vertex> param, std::shared_ptr<vertex> grad,
            std::shared_ptr<vertex> opt_data, double momentum, double step_size)
        {
            auto param_order = leaves_pre_order(param);
            auto grad_order = leaves_pre_order(grad);
            auto opt_data_order = leaves_pre_order(opt_data);

            assert(param_order.size() == grad_order.size()
                && grad_order.size() == opt_data_order.size());

            for (int i = 0; i < param_order.size(); ++i) {
                if (grad_order[i]->data == nullptr) {
                    continue;
                }

                if (param_order[i]->type == "tensor") {
                    opt::const_step_update_momentum(
                        get_tensor(param_order[i]),
                        get_tensor(grad_order[i]),
                        get_tensor(opt_data_order[i]),
                        momentum, step_size);
                } else if (param_order[i]->type == "gpu-tensor") {
                    opt::gpu::const_step_update_momentum(
                        get_gpu_tensor(param_order[i]),
                        get_gpu_tensor(grad_order[i]),
                        get_gpu_tensor(opt_data_order[i]),
                        momentum, step_size);
                }
            }
        }

        void adagrad_update(std::shared_ptr<vertex> param, std::shared_ptr<vertex> grad,
            std::shared_ptr<vertex> accu_grad_sq, double step_size)
        {
            auto param_order = leaves_pre_order(param);
            auto grad_order = leaves_pre_order(grad);
            auto accu_grad_sq_order = leaves_pre_order(accu_grad_sq);

            assert(param_order.size() == grad_order.size()
                && grad_order.size() == accu_grad_sq_order.size());

            for (int i = 0; i < param_order.size(); ++i) {
                if (grad_order[i]->data == nullptr) {
                    continue;
                }

                if (param_order[i]->type == "tensor") {
                    opt::adagrad_update(
                        get_tensor(param_order[i]),
                        get_tensor(grad_order[i]),
                        get_tensor(accu_grad_sq_order[i]),
                        step_size);
                } else if (param_order[i]->type == "gpu-tensor") {
                    opt::gpu::adagrad_update(
                        get_gpu_tensor(param_order[i]),
                        get_gpu_tensor(grad_order[i]),
                        get_gpu_tensor(accu_grad_sq_order[i]),
                        step_size);
                }
            }
        }

        void rmsprop_update(std::shared_ptr<vertex> param, std::shared_ptr<vertex> grad,
            std::shared_ptr<vertex> opt_data, double decay, double step_size)
        {
            auto param_order = leaves_pre_order(param);
            auto grad_order = leaves_pre_order(grad);
            auto opt_data_order = leaves_pre_order(opt_data);

            assert(param_order.size() == grad_order.size()
                && grad_order.size() == opt_data_order.size());

            for (int i = 0; i < param_order.size(); ++i) {
                if (grad_order[i]->data == nullptr) {
                    continue;
                }

                if (param_order[i]->type == "tensor") {
                    opt::rmsprop_update(
                        get_tensor(param_order[i]),
                        get_tensor(grad_order[i]),
                        get_tensor(opt_data_order[i]),
                        decay, step_size);
                } else if (param_order[i]->type == "gpu-tensor") {
                    opt::gpu::rmsprop_update(
                        get_gpu_tensor(param_order[i]),
                        get_gpu_tensor(grad_order[i]),
                        get_gpu_tensor(opt_data_order[i]),
                        decay, step_size);
                }
            }
        }

        void adam_update(std::shared_ptr<vertex> param,
            std::shared_ptr<vertex> grad,
            std::shared_ptr<vertex> first_moment,
            std::shared_ptr<vertex> second_moment,
            int& time, double alpha, double beta1, double beta2)
        {
            auto param_order = leaves_pre_order(param);
            auto grad_order = leaves_pre_order(grad);
            auto first_moment_order = leaves_pre_order(first_moment);
            auto second_moment_order = leaves_pre_order(second_moment);

            assert(param_order.size() == grad_order.size()
                && grad_order.size() == first_moment_order.size()
                && first_moment_order.size() == second_moment_order.size());

            for (int i = 0; i < param_order.size(); ++i) {
                if (grad_order[i]->data == nullptr) {
                    continue;
                }

                if (param_order[i]->type == "tensor") {
                    opt::adam_update(
                        get_tensor(param_order[i]),
                        get_tensor(grad_order[i]),
                        get_tensor(first_moment_order[i]),
                        get_tensor(second_moment_order[i]),
                        time, alpha, beta1, beta2);
                } else if (param_order[i]->type == "gpu-tensor") {
                    opt::gpu::adam_update(
                        get_gpu_tensor(param_order[i]),
                        get_gpu_tensor(grad_order[i]),
                        get_gpu_tensor(first_moment_order[i]),
                        get_gpu_tensor(second_moment_order[i]),
                        time, alpha, beta1, beta2);
                }
            }
        }

        void const_step_opt::update(std::shared_ptr<vertex> grad)
        {
            tensor_tree::gpu::const_step_update(param, grad, step_size);
        }

        void const_step_momentum_opt::update(std::shared_ptr<vertex> grad)
        {
            tensor_tree::gpu::const_step_update_momentum(param, grad, opt_data, momentum, step_size);
        }

        void adagrad_opt::update(std::shared_ptr<vertex> grad)
        {
            tensor_tree::gpu::adagrad_update(param, grad, accu_grad_sq, step_size);
        }

        void rmsprop_opt::update(std::shared_ptr<vertex> grad)
        {
            tensor_tree::gpu::rmsprop_update(param, grad, accu_grad_sq, decay, step_size);
        }

        void adam_opt::update(std::shared_ptr<vertex> grad)
        {
            tensor_tree::gpu::adam_update(param, grad, first_moment, second_moment,
                time, alpha, beta1, beta2);
        }

    }

}
